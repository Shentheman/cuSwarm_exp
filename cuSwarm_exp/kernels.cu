#include "hip/hip_runtime.h"
#include "kernels.cuh"

/*********************
***** VARIABLES ******
*********************/

// Device pointers for simulation agent data arrays
/// This is the back end of d_vbo_resource on device
/// (float x, float y, float z, float color)
float4* d_positions;
/// (float x, float y, float speed)
float3* d_velocities;
int* d_modes;
int* d_leaders;
int* d_nearest_leader;
uint* d_leader_countdown;
int4* d_laplacian;
bool* d_ap;
/// The positions of all the obstacles
/// (float x, float y, int flag, float color) (we replace z with flag)
float4* d_positions_obs;
/* front end = the array on the device which connected to the VBO on the host
 * back end = the array on the device used for updating the VBO on the host
 * The reason to have both front and back is to have the back end save data
 * from the past and update the front end in each step. Without the back end,
 * in each step, the front end will be updated entirely.
 */
// The front end buffer which connects to vbo_resource in the host
float4* d_vbo_resource;

// Device pointers for environment variables
bool* d_occupancy;
float2* d_flow_pos;
float2* d_flow_dir;

// CUDA random states
hiprandState* d_rand_states;

// Kernel launch parameters
uint grid_dim;
dim3 block;
dim3 grid;

// CUDA streams
hipStream_t streams[4];

/****************************
***** HELPER FUNCTIONS ******
****************************/

void cudaAllocate(Parameters p) {

  // Allocate space on device for simulation arrays
  hipMalloc(&d_positions, p.num_robots*sizeof(float4));
  hipMemset(d_positions, 0, p.num_robots*sizeof(float4));

  hipMalloc(&d_velocities, p.num_robots * sizeof(float3));
  hipMemset(d_velocities, 0, p.num_robots*sizeof(float3));

  hipMalloc(&d_modes, p.num_robots * sizeof(int));
  hipMemset(d_modes, 0, p.num_robots*sizeof(int));

  hipMalloc(&d_leaders, p.num_robots * sizeof(int));
  hipMemset(d_leaders, 0, p.num_robots*sizeof(int));

  hipMalloc(&d_nearest_leader, p.num_robots * sizeof(int));
  hipMemset(d_nearest_leader, 0, p.num_robots*sizeof(int));

  hipMalloc(&d_leader_countdown, p.num_robots * sizeof(uint));
  hipMemset(d_leader_countdown, 0, p.num_robots*sizeof(uint));

  hipMalloc(&d_laplacian, p.num_robots*p.num_robots*sizeof(int4));
  hipMemset(d_laplacian, 0, p.num_robots*p.num_robots*sizeof(int4));

  hipMalloc(&d_ap, p.num_robots * sizeof(bool));
  hipMemset(d_ap, 0, p.num_robots*sizeof(bool));

  // Allocate space on device for environment arrays
  hipMalloc(&d_occupancy, p.world_size*p.world_size*10*10*sizeof(bool));
  hipMemset(d_occupancy, 0, p.world_size*p.world_size*10*10*sizeof(bool));

  hipMalloc(&d_flow_pos, 256*sizeof(float2));
  hipMemset(d_flow_pos, 0, 256*sizeof(float2));

  hipMalloc(&d_flow_dir, 256*sizeof(float2));
  hipMemset(d_flow_dir, 0, 256*sizeof(float2));

  // Allocate space on device for random state variables
  hipMalloc(&d_rand_states, p.num_robots * sizeof(hiprandState));
  hipMemset(d_rand_states, 0, p.num_robots * sizeof(hiprandState));

  /// For each robot, we will have NUM_ANGLE_RAY_TRACE possible obstacles
  /// XXX: we need to use double pointer in cuda
  /// https://stackoverflow.com/questions/7989039/use-of-cudamalloc-why-the-double-pointer
  hipMalloc(&d_positions_obs, 
      p.num_robots*NUM_ANGLE_RAY_TRACE*sizeof(float4));
  hipMemset(d_positions_obs, 0, 
      p.num_robots*NUM_ANGLE_RAY_TRACE*sizeof(float4));

  hipMalloc(&d_vbo_resource, p.num_robots*sizeof(float4));
  hipMemset(d_vbo_resource, 0, p.num_robots*sizeof(float4));

  // Set kernel launch parameters
  grid_dim = (uint)(ceilf((float)(p.num_robots) / BLOCK_SIZE));
  block = dim3(min(p.num_robots, BLOCK_SIZE), 1, 1);
  grid = dim3(grid_dim, 1, 1);

  // Create streams for simultaneous kernel launches
  hipStreamCreate(&streams[0]);
  hipStreamCreate(&streams[1]);
  hipStreamCreate(&streams[2]);
  hipStreamCreate(&streams[3]);
}

void cuFree()
{
  // Free arrays from simulation device memory
  hipFree(d_positions);
  hipFree(d_velocities);
  hipFree(d_modes);
  hipFree(d_leaders);
  hipFree(d_nearest_leader);
  hipFree(d_leader_countdown);
  hipFree(d_laplacian);
  hipFree(d_occupancy);
  hipFree(d_flow_pos);
  hipFree(d_flow_dir);
  hipFree(d_rand_states);

  hipFree(d_positions_obs);
  hipFree(d_vbo_resource);

  // Delete streams for simultaneous kernel launches
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);
  hipStreamDestroy(streams[2]);
  hipStreamDestroy(streams[3]);
}

void launchInitKernel(Parameters p) {

  // Run initialization kernel to load initial simulation state
  init_kernel <<<grid, block>>>(d_positions, d_velocities, d_modes, 
      d_rand_states, (ulong)(time(NULL)), d_flow_pos, d_flow_dir, 
      d_nearest_leader, d_leader_countdown, p, d_positions_obs);
}

void launchInitKernel(Parameters p, 
        struct hipGraphicsResource **vbo_resource) {

  // Map 1 OpenGL buffer object (resource) for writing from CUDA 
  //   with 0 stream for synchronization
  int error = hipGraphicsMapResources(1, vbo_resource, 0);
  if (error != 0) printf("ERROR launchInitKernel 1 = %d\n",error);
  size_t num_bytes;
  // Get an device pointer through which to access a mapped graphics resource.
  error = hipGraphicsResourceGetMappedPointer((void **)&d_vbo_resource, 
        &num_bytes, *vbo_resource);
  if (error != 0) printf("ERROR launchInitKernel 2 = %d\n",error);

  // Run initialization kernel to load initial simulation state
  launchInitKernel(p);

  /// After update the back end arrays, update the front end arrays
  /// so that the host can draw the new VBO
  /*http://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g3f51e3575c2178246db0a94a430e0038*/
  error = hipMemcpy(d_vbo_resource, d_positions, 
          p.num_robots*sizeof(float4), hipMemcpyDefault);
  if (error != 0) printf("ERROR launchInitKernel 3 = %d\n",error);
  /// XXX: we cannot add p.number_robots*sizeof(float4) - pointer arithmatic
  /*error = hipMemcpy(d_vbo_resource+p.num_robots, d_positions_obs, */
          /*GRID_SIZE*sizeof(float4), hipMemcpyDefault);*/
  /*if (error != 0) printf("ERROR launchInitKernel 4 = %d\n",error);*/

  // Unmap OpenGL buffer object
  error = hipGraphicsUnmapResources(1, vbo_resource, 0);
  if (error != 0) printf("ERROR launchInitKernel 5 = %d\n",error);
}

void launchMainKernel(float3 gh, float2 gp, uint sn, int* leaders, bool* ap, 
  Parameters p) {

  // Copy leader and articulation point data to GPU
  int error = hipMemcpy(d_leaders, leaders, p.num_robots * sizeof(int),
    hipMemcpyHostToDevice);
  if (error != 0) printf("ERROR launchMainKernel 1st = %d\n",error);
  error = hipMemcpy(d_ap, ap, p.num_robots * sizeof(bool), hipMemcpyHostToDevice);
  if (error != 0) printf("ERROR launchMainKernel 2nd = %d\n",error);
  
  /// XXX: Now we are still in host, not on device
  /// So we cannot access nor print d_positions_obs

  // Launch the main and side kernels
  main_kernel <<<grid, block, 0, streams[0]>>>(d_positions, d_velocities, 
      d_modes, gh, gp,  d_rand_states, d_ap, d_flow_pos, d_flow_dir, 
      d_occupancy, p, sn, d_positions_obs);

  // Run side kernel for extra computations outside the control loop
  side_kernel <<<grid, block, 0, streams[1]>>>(d_positions, d_modes, 
      d_leaders, d_rand_states, p, d_nearest_leader, d_leader_countdown, 
      d_laplacian, sn);

  // Synchronize kernels on device
  hipDeviceSynchronize();
}

void launchMainKernel(float3 gh, float2 gp, uint sn, int* leaders, bool* ap, 
  Parameters p, struct hipGraphicsResource **vbo_resource) {

  // Map OpenGL buffer object for writing from CUDA
  int error = hipGraphicsMapResources(1, vbo_resource, 0);
  if (error != 0) printf("ERROR launchMainKernel 1 = %d\n",error);
  size_t num_bytes;
  error = hipGraphicsResourceGetMappedPointer((void **)&d_vbo_resource, 
      &num_bytes, *vbo_resource);
  if (error != 0) printf("ERROR launchMainKernel 2 = %d\n",error);

  launchMainKernel(gh, gp, sn, leaders, ap, p);

  error = hipMemcpy(d_vbo_resource, d_positions, 
          p.num_robots*sizeof(float4), hipMemcpyDefault);
  if (error != 0) printf("ERROR launchMainKernel 3 = %d\n",error);
  /*error = hipMemcpy(d_vbo_resource+p.num_robots, d_positions_obs, */
          /*GRID_SIZE*sizeof(float4), hipMemcpyDefault);*/
  /*if (error != 0) printf("ERROR launchMainKernel 4 = %d\n",error);*/

  // Unmap OpenGL buffer object
  error = hipGraphicsUnmapResources(1, vbo_resource, 0);
  if (error != 0) printf("ERROR launchMainKernel 5 = %d\n",error);
}

void getData(uint num_robots, float4* positions, float3* velocities, int* modes,
    float4* positions_obs) {

  // Copy simulation data from device to host arrays
  hipMemcpy(positions, d_positions, 
      num_robots*sizeof(float4), hipMemcpyDeviceToHost);
  hipMemcpy(velocities, d_velocities, 
      num_robots*sizeof(float3), hipMemcpyDeviceToHost);
  hipMemcpy(modes, d_modes, 
      num_robots*sizeof(int), hipMemcpyDeviceToHost);
  /// the positions of all the obstacles
  hipMemcpy(positions_obs, d_positions_obs, 
      num_robots*NUM_ANGLE_RAY_TRACE*sizeof(float4), hipMemcpyDeviceToHost);
}

void getData(uint num_robots, uint n_positions_obs, float4* positions, 
    float3* velocities, int* modes, int* nearest_leader, 
    uint* leader_countdown, float4* positions_obs) {
  // Copy simulation data from device to host arrays
  hipMemcpy(positions, d_positions, 
      num_robots*sizeof(float4), hipMemcpyDeviceToHost);
  hipMemcpy(velocities, d_velocities, 
      num_robots*sizeof(float3), hipMemcpyDeviceToHost);
  hipMemcpy(modes, d_modes, 
      num_robots*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(nearest_leader, d_nearest_leader, 
      num_robots*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(leader_countdown, d_leader_countdown, 
      num_robots*sizeof(uint), hipMemcpyDeviceToHost);
  /// the positions of all the obstacles
  hipMemcpy(positions_obs, d_positions_obs, 
      num_robots*NUM_ANGLE_RAY_TRACE*sizeof(float4), hipMemcpyDeviceToHost);
}

void getLaplacian(uint n, int4* laplacian) {
  hipMemcpy(laplacian, d_laplacian, n * n * sizeof(int4), hipMemcpyDeviceToHost);
}

void setData(uint n, float4* positions, float3* velocities, int* modes)
{
  // Copy simulation data from host to device arrays
  hipMemcpy(d_positions, positions, n * sizeof(float4), hipMemcpyHostToDevice);
  hipMemcpy(d_velocities, velocities, n * sizeof(float3), hipMemcpyHostToDevice);
  hipMemcpy(d_modes, modes, n * sizeof(int), hipMemcpyHostToDevice);
}

void setData(uint n, float4* positions, float3* velocities, int* modes,
  int* nearest_leader, uint* leader_countdown)
{
  // Copy simulation data from host to device arrays
  hipMemcpy(d_positions, positions, n * sizeof(float4), hipMemcpyHostToDevice);
  hipMemcpy(d_velocities, velocities, n * sizeof(float3), hipMemcpyHostToDevice);
  hipMemcpy(d_modes, modes, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_nearest_leader, nearest_leader, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_leader_countdown, leader_countdown, n * sizeof(uint), hipMemcpyHostToDevice);
}

void setOccupancy(Parameters p, bool* occupancy) {
  // Copy occupancy data from host to device array
  hipMemcpy(d_occupancy, occupancy, 
      p.world_size*p.world_size*10*10*sizeof(bool), hipMemcpyHostToDevice);
}

/**************************
***** CUDA FUNCTIONS ******
**************************/

__global__ void init_kernel(float4* pos, float3* vel, int* mode, 
    hiprandState* rand_state, ulong seed, float2* flow_pos, float2* flow_dir, 
    int* nearest_leader, uint* leader_countdown, Parameters p,
    float4* pos_obs) {

  // Index of this robot
  uint i = blockIdx.x * blockDim.x + threadIdx.x;

  /*Just let one robot to do it*/
  if (i == 0) {
      for (uint j = 0; j < p.num_robots*NUM_ANGLE_RAY_TRACE; j++) {
        pos_obs[j] = make_float4(-p.world_size, -p.world_size,
            GRID_UNEXPLORED, 0.0f);
      }
      for (uint j = 0; j < p.num_robots*NUM_ANGLE_RAY_TRACE; j++) {
        printf("array[%d]=(%f,%f,%f,%f)",j, pos_obs[j].x, pos_obs[j].y, 
            pos_obs[j].z, pos_obs[j].w);
      }
  }

  __syncthreads();

  // Frequently-used parameters
  float n_f = (float)(p.num_robots);
  float ws = (float)(p.world_size);

  // Seed the RNG
  hiprand_init(seed, i, 0, &rand_state[i]);
  hiprandState local_state = rand_state[i];

  // Initialize mode
  // Initially, there are no leaders
  mode[i] = p.hops + 1;
  // Make the first noise % robots have a mode of -1 (noise mode)
  if (i < (int)(p.noise * n_f)) {
    mode[i] = MODE_NOISE;
  }
  
  // Initialize nearest_leader and leader_countdown arrays 
  // for RCC leader selection
  // Initially, there are no leaders
  nearest_leader[i] = LEADER_NON_EXIST;
  leader_countdown[i] = i;

  // Randomly place this robot within the starting circle
  float theta = hiprand_uniform(&local_state) * 2.0f * PI;
  float unit_r = hiprand_uniform(&local_state);
  float sqrt_unit_r = sqrtf(unit_r);
  /*10,0, 0.801, 2.88*/
  /*printf ("Robot %d Initial pos = %f, %f, %f\n", i, p.start_size, sqrt_unit_r, theta);*/
  float x_pos = p.start_size * sqrt_unit_r * cosf(theta);
  float y_pos = p.start_size * sqrt_unit_r * sinf(theta);

  // Set the initial color
  Color color;
  /// whether this robot member encounters obstacles
  bool is_obs_encountered = false;
  setColorSwarm(&(color.components), 1, false, i, p, is_obs_encountered);

  // Set speed manually from params.txt
  float speed = p.vel_bound / 60.0f;
  // Set initial position, color and velocity
  pos[i] = make_float4(x_pos, y_pos, 0.0f, color.c);
  vel[i] = make_float3(0.0f, 0.0f, speed);

  // Assign a random flow to 256 points in the environment
  if (i < 256) {
    // First, set flow position
    float x_flow_pos = (hiprand_uniform(&local_state) * ws) - (ws / 2.0f);
    float y_flow_pos = (hiprand_uniform(&local_state) * ws) - (ws / 2.0f);
    flow_pos[i] = make_float2(x_flow_pos, y_flow_pos);

    // Second, set flow direction
    float x_flow_dir = (hiprand_uniform(&local_state) * 2.0f) - 1.0f;
    float y_flow_dir = (hiprand_uniform(&local_state) * 2.0f) - 1.0f;
    flow_dir[i] = make_float2(x_flow_dir, y_flow_dir);
  }
}

__global__ void side_kernel(float4* pos, int* mode, int* leaders, 
        hiprandState* rand_state, Parameters p, int* nearest_leader, 
        uint* leader_countdown, int4* laplacian, uint sn) {

  // Index of this robot
  uint i = blockIdx.x * blockDim.x + threadIdx.x;

  // Do not perform any leader calculation if a noise robot, 
  // or in a non-update step
  if (mode[i] != MODE_NOISE) {
    // Perform either RCC or CH leader assignment, depending on parameter

    /// I. Random Competition based Clustering (RCC) leader selection
    if (p.leader_selection == 0) {

      // Holds the new mode and nearest leader during computation
      int new_mode = mode[i];
      int new_nearest_leader = nearest_leader[i];

      // If the leader countdown for this robot has expired,
      // switch leader state and reset the timer;
      if (leader_countdown[i] == 0) {
        // Switch to a leader if not already; else switch to non-leader

        /// Wait and check whether other robots claim leadership
        if (mode[i] == MODE_LEADER) {
          // Assign it as non-leader for 1 second
          new_mode = MODE_NON_LEADER_MAX;
          new_nearest_leader = LEADER_NON_EXIST;
          leader_countdown[i] = 60;
        } 
        // (1) If when the timer ends, the robot i has still not been assigned 
        //     a leader, which means that none of other robots 
        //     have claimed as leaders, then this robot i will claim to be 
        //     the leader. 
        else if (mode[i] > MODE_LEADER) {
          // Assign it as a leader for 5 seconds
          new_mode = MODE_LEADER;
          new_nearest_leader = i;
          leader_countdown[i] = 360;
        }
      }
      // (2) If before the timer ends, the robot i was assigned a leader,
      //     then let robot i follow this leader and reset the timer.
      //
      //     i.e. before the timer ends, one robot A != i, 
      //     claims to be a leader,
      //     then A's mode becomes 0,
      //     A's neighbors' modes becomes 1,
      //     A's neighbors' neighbors' modes becomes 2, 
      //     ......
      //     the last neighbor's mode becomes p.hops-1.
      //
      //     Now the robot i's mode >= p.hops which means that 
      //     robot i does not have any leaders now,
      //     because all the other leaders are too far away from robot i.
      //     So now we can finally assign A as the leader of this robot i.
      else {
        // Iterate through all neighbor robots
        for (int n = 0; n < p.num_robots; n++) {
          // Get the distance between the robots
          float4 me = pos[i];
          float4 them = pos[n];
          float2 dist = make_float2(me.x - them.x, me.y - them.y);
          // Range for hops/leader calculations is max_range / 2
          bool within_range = euclidean(dist) < (p.range_l);

          // Peform operation based on if the robots are within range
          if (within_range && i != n) {
            // If a neighbor with a hop value less than the max and 
            // not greater than this robot is found with a lower ID, 
            // set this robot to non-leader status
            if ((mode[n] < p.hops) && (mode[n] <= mode[i]) && (n < i))
            {
              new_mode = mode[n] + 1;
              new_nearest_leader = nearest_leader[n];
              // Reset leader countdown timer to 1 second
              leader_countdown[i] = 60;
            }
          }
        }
      }

      // Synchronize threads before updating mode and neighbor arrays
      __syncthreads();

      // Update this robot's mode and nearest leader
      mode[i] = new_mode;
      nearest_leader[i] = new_nearest_leader;

      // Decrease countdown timer
      leader_countdown[i]--;
    }
    /// II. Convex Hull leader assignment
    else if (p.leader_selection == 1) {
      // Look at the index of this robot in the leader list to determine if 
      // this robot should be a leader
      if (leaders[i] == 0) {
        mode[i] = 0;
      }
      else {
        mode[i] = 1;
      }
    }
  }

  // Degree for laplacian
  uint degree_a = 0, degree_b = 0, degree_c = 0, degree_d = 0;
  // Iterate through all other robots
  for (uint j = 0; j < p.num_robots; j++) {
    if (i != j) {
      // Get the distance between the robots
      float4 me = pos[i];
      float4 them = pos[j];
      float2 dist_xy = make_float2(me.x - them.x, me.y - them.y);
      float dist = euclidean(dist_xy);
      // Set the non-diagonal values based on whether robots are connected
      // at the four different ranges, -1 means connected, 0 disconnected
      if (dist < p.range) {
        degree_a++;
        laplacian[(i * p.num_robots) + j].x = LAPLACIAN_CONNECTED;
        laplacian[(j * p.num_robots) + i].x = LAPLACIAN_CONNECTED;
      }
      else {
        laplacian[(i * p.num_robots) + j].x = LAPLACIAN_DISCONNECTED;
        laplacian[(j * p.num_robots) + i].x = LAPLACIAN_DISCONNECTED;
      }
      if (dist < p.range_r) {
        degree_b++;
        laplacian[(i * p.num_robots) + j].y = LAPLACIAN_CONNECTED;
        laplacian[(j * p.num_robots) + i].y = LAPLACIAN_CONNECTED;
      }
      else {
        laplacian[(i * p.num_robots) + j].y = LAPLACIAN_DISCONNECTED;
        laplacian[(j * p.num_robots) + i].y = LAPLACIAN_DISCONNECTED;
      }
      if (dist < p.range_f) {
        degree_c++;
        laplacian[(i * p.num_robots) + j].z = LAPLACIAN_CONNECTED;
        laplacian[(j * p.num_robots) + i].z = LAPLACIAN_CONNECTED;
      }
      else {
        laplacian[(i * p.num_robots) + j].z = LAPLACIAN_DISCONNECTED;
        laplacian[(j * p.num_robots) + i].z = LAPLACIAN_DISCONNECTED;
      }
      if (dist < p.range_l) {
        degree_d++;
        laplacian[(i * p.num_robots) + j].w = LAPLACIAN_CONNECTED;
        laplacian[(j * p.num_robots) + i].w = LAPLACIAN_CONNECTED;
      }
      else {
        laplacian[(i * p.num_robots) + j].w = LAPLACIAN_DISCONNECTED;
        laplacian[(j * p.num_robots) + i].w = LAPLACIAN_DISCONNECTED;
      }
    }
  }
  // Set the diagonal of the laplacian to the degree of corresponding robot
  laplacian[(i * p.num_robots) + i].x = degree_a;
  laplacian[(i * p.num_robots) + i].y = degree_b;
  laplacian[(i * p.num_robots) + i].z = degree_c;
  laplacian[(i * p.num_robots) + i].w = degree_d;
}

__global__ void main_kernel(float4* pos, float3* vel, int* mode, 
  float3 goal_heading, float2 goal_point, hiprandState* rand_state, 
  bool* ap, float2* flow_pos, float2* flow_dir, bool* occupancy, 
  Parameters p, uint sn,
  float4* pos_obs) {

  // Index of this robot
  uint i = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float4 s_pos[BLOCK_SIZE];
  __shared__ float3 s_vel[BLOCK_SIZE];
  __shared__ int s_mode[BLOCK_SIZE];
  __shared__ bool s_ap[BLOCK_SIZE];

  __syncthreads();

  // Variables for this robot's data
  float4 myPos = pos[i];
  int myMode = mode[i];
  float mySpeed = p.vel_bound / 60.0f;
  float dist_to_obstacle = p.range_o;
  hiprandState local_state = rand_state[i];
  /// whether this robot member encounters obstalces
  bool is_obs_encountered = false;

  // Computation variable initializations
  float2 min_bounds = make_float2(0.0f, 0.0f);
  float2 max_bounds = make_float2(0.0f, 0.0f);
  float2 repel = make_float2(0.0f, 0.0f);
  float2 align = make_float2(0.0f, 0.0f);
  float2 cohere = make_float2(0.0f, 0.0f);
  float2 avoid = make_float2(0.0f, 0.0f);
  float2 flow = make_float2(0.0f, 0.0f);
  float2 goal = make_float2(0.0f, 0.0f);

  // Ignore behavior operations if this robot is noise
  if (myMode != MODE_NOISE) {
    // If we are flocking and a leader, set the alignment vector towards 
    // the goal point
    if (p.behavior == BEHAVIOR_FLOCKING && myMode == MODE_LEADER) {
      align.x = goal_heading.x;
      align.y = goal_heading.y;
    }
    /// Followers will only follow the leaders 100% without having a align vector

    // Iterate through blocks to use shared memory within a block
    for (uint tile = 0; tile < gridDim.x; tile++) {

      // Assign shared memory for this block
      uint n = tile * blockDim.x + threadIdx.x;
      s_pos[threadIdx.x] = pos[n];
      s_vel[threadIdx.x] = vel[n];
      s_mode[threadIdx.x] = mode[n];
      s_ap[threadIdx.x] = ap[n];

      // Synchronize threads after shared memory is assigned
      __syncthreads();

      // Iterate through all threads in this block
      for (uint ti = 0; ti < blockDim.x; ti++) {
        // Do not perform an interaction between this robot and itself
        if (i != tile * blockDim.x + ti) {

          // Calculate the distance between the two robots on all axes
          float dist_x = s_pos[ti].x - myPos.x;
          float dist_y = s_pos[ti].y - myPos.y;

          // Calculate the Euclidean distance between the two robots
          float dist = euclidean(make_float2(dist_x, dist_y));

          // Perform interaction for neighbors within range
          if (dist <= p.range) {

            // Create collecte distance variable (for readability)
            float3 dist3 = make_float3(dist_x, dist_y, dist);

            // Perform the interaction for this robot pair based on 
            // the current behavior
            switch (p.behavior) {
            case 0:
              rendezvous(dist3, &min_bounds, &max_bounds, &repel, s_ap[ti], p);
              break;
            case 1:
              flock(myMode, s_vel[ti], s_mode[ti], dist3, &repel, &align, &cohere, s_ap[ti], p);
              break;
            case 2:
              disperse(dist3, &repel, &cohere, s_ap[ti], p);
              break;
            case 3:
              rendezvousToPoint(dist3, &repel, p);
              break;
            }
          }
        }
      }
    }

    /// XXX: We can save the positions of all the obstacles 
    /// detected by this robot member
    // Perform obstacle avoidance computation for this robot
    obstacleAvoidance(myPos, &avoid, &dist_to_obstacle, occupancy, p, 
        pos_obs, i, is_obs_encountered);

    // Finish necessary summary computations for each behavior
    switch (p.behavior) {
      case 0:
        // Finish computation of parallel circumcenter algorithm
        cohere.x = ((min_bounds.x + max_bounds.x) / 2.0f);
        cohere.y = ((min_bounds.y + max_bounds.y) / 2.0f);
        break;
      case 1:
        break;
      case 2:
        break;
      case 3:
        // Set align vector to point toward goal point
        float align_angle = atan2f(goal_point.y - myPos.y, goal_point.x - myPos.x);
        align.x = cosf(align_angle);
        align.y = sinf(align_angle);
        break;
    }

    // If velocity is affected by random flows, calculate flow effect here
    if (p.current > 0.0f) {
      for (uint fi = 0; fi < 256; fi++) {
        float2 dist_v = make_float2(myPos.x - flow_pos[fi].x, myPos.y - flow_pos[fi].y);
        float dist = euclidean(dist_v);
        flow.x += flow_dir[fi].x * (1.0f / dist);
        flow.y += flow_dir[fi].y * (1.0f / dist);
      }
    }

    // Scale all component vectors to their weights and compute goal vector
    rescale(&repel, p.repel_weight, false);
    rescale(&align, p.align_weight, false);
    rescale(&cohere, p.cohere_weight, false);
    rescale(&avoid, 4.0f * powf((p.range_o - dist_to_obstacle), 4.0f), false);
    // Add random currents, if applicable
    if (p.current > 0.0f) {
      rescale(&flow, p.current, false);
    }

    // Combine behavior components to make the new goal vector
    goal.x = repel.x + align.x + cohere.x + avoid.x + flow.x;
    goal.y = repel.y + align.y + cohere.y + avoid.y + flow.y;
  }
  else if (myMode == MODE_NOISE) { // Noise robots
    // Apply error from the normal distribution to the velocity
    goal.x = hiprand_normal(&local_state);
    goal.y = hiprand_normal(&local_state);
  }

  // Cap the angular velocity
  capAngularVelocity(make_float2(vel[i].x, vel[i].y), &goal, p.ang_bound / 60.0f);
  // Rescale the goal to the calculated speed
  rescale(&goal, mySpeed, true);

  // Synchronize threads before updating robot state variables
  __syncthreads();

  // Set the color based on current mode
  Color color;
  setColorSwarm(&(color.components), myMode, ap[i], i, p, is_obs_encountered);
  // Update velocity and mode
  vel[i] = make_float3(goal.x, goal.y, mySpeed);

  // Update position
  pos[i] = make_float4(myPos.x + vel[i].x, myPos.y + vel[i].y, 0.0f, color.c);

  ////https://stackoverflow.com/questions/21005845/how-to-get-float-bytes
  ////https://stackoverflow.com/questions/920511/how-to-visualize-bytes-with-c-c

  /*char colorBytes[sizeof(float)];*/
  /*memcpy(colorBytes, &color.c, sizeof(float));*/
  /*printf ("color = [%02x, %02x, %02x, %02x] (%d,%d,%d)\n", */
          /*colorBytes[0], colorBytes[1], colorBytes[2], colorBytes[3],*/
          /*color.components.x, color.components.y, color.components.z,*/
          /*color.components.w);*/

  // Update random state for CUDA RNG
  rand_state[i] = local_state;
}

__device__ void rendezvous(float3 dist3, float2* min_bounds, float2* max_bounds, 
  float2* repel, bool is_ap, Parameters p)
{
  if (dist3.z <= p.range_r) {
    // REPEL
    // Repel from robots within repel range
    float weight = powf(p.range_r - dist3.z, 2.0f);
    repel->x -= weight * dist3.x;
    repel->y -= weight * dist3.y;
  }
  if (dist3.z <= p.range && dist3.z > p.range_r)
  {
    // COHERE
    // Robots cohere to the center of the rectangle that bounds neighbors
    // Do not cohere to neighbors within repel range
    min_bounds->x = fminf(min_bounds->x, dist3.x);
    min_bounds->y = fminf(min_bounds->y, dist3.y);
    max_bounds->x = fmaxf(max_bounds->x, dist3.x);
    max_bounds->y = fmaxf(max_bounds->y, dist3.y);
  }
}

__device__ void flock(int myMode, float3 nVel, int nMode, float3 dist3,
  float2* repel, float2* align, float2* cohere, bool is_ap, Parameters p) {

  // Main flocking section
  if (dist3.z <= p.range_f) {
    // REPEL
    // Robots repel from neighbors within flocking repel range
    float weight = powf(p.range_f - dist3.z, 2.0f);
    repel->x -= weight * dist3.x;
    repel->y -= weight * dist3.y;
  }
  if (myMode != 0) {
    // ALIGN
    float weight;
    (nMode == 0) ? weight = 10.0f : weight = 1.0f;
    align->x += weight * nVel.x;
    align->y += weight * nVel.y;
  }
  if (dist3.z < p.range && dist3.z > p.range_f) {
    // COHERE
    // Do not cohere to neighbors within repel range
    float weight = powf(dist3.z - p.range_f, 2.0f);
    cohere->x += weight * dist3.x;
    cohere->y += weight * dist3.y;
  }
}

__device__ void disperse(float3 dist3, float2* repel, float2* cohere, bool is_ap, 
  Parameters p) {

  // Determine whether we should repel or cohere based on the 
  // distance to the neighbor
  if (dist3.z <= p.range_d) {
    // REPEL
    // Robots repel from neighbors within disperse repel range
    float weight = powf(p.range_d - dist3.z, 2.0f);
    repel->x -= weight * dist3.x;
    repel->y -= weight * dist3.y;
  }
  if (dist3.z <= p.range && dist3.z > p.range_d) {
    // COHERE
    // Do not cohere to robots within disperse repel range
    float weight = 0.0f;// powf(dist3.z - p.range_d, 3.0f);
    cohere->x += weight * dist3.x;
    cohere->y += weight * dist3.y;
  }
}

__device__ void rendezvousToPoint(float3 dist3, float2* repel, Parameters p) {

  if (dist3.z <= p.range_r) {
    // REPEL
    // Repel from robots within repel range
    float weight = powf(p.range_r - dist3.z, 2.0f);
    repel->x -= weight * dist3.x;
    repel->y -= weight * dist3.y;
  }
}

__device__ void obstacleAvoidance(float4 myPos, float2* avoid, 
  float* dist_to_obstacle, bool* occupancy, Parameters p, 
  float4* pos_obs, uint robot_index, bool &is_obs_encountered) {

  /// Checks the collision of one robot member with the obstacles in the map 
  /// and the borders of the world.
  /// But it does not check the collision of the member with other robot members.

  *(dist_to_obstacle) = FLT_MAX;
  int counter = 0;
  for (float i = 0; i < 2.0f * PI; i += RAY_TRACE_INTERVAL) {
    float cos = cosf(i);
    float sin = sinf(i);
    // Ray trace along this angle up to the robot's avoidance range
    for (float r = 0.0f; r < p.range_o; r += 1.0f) {
      float x_check = myPos.x + r * cos;
      float y_check = myPos.y + r * sin;

      int occupancy_ind = occupancySub2Ind(x_check, y_check, p);
      bool occupied = false;
      if (occupancy_ind == -1)
        occupied = true;
      else
        occupied = occupancy[occupancy_ind];

      if (occupied == true) {
        // Get weight for obstacle repulsion force
        float weight = powf(1.0f - (r / p.range_o), 2.0f);
        // Update the distance to the closest obstacle
        if (r < *dist_to_obstacle) {
          *dist_to_obstacle = r;
        }
        // Update the obstacle avoidance vector
        avoid->x += weight * -r * cos;
        avoid->y += weight * -r * sin;

        int tmp = robot_index*NUM_ANGLE_RAY_TRACE+counter;
        counter ++;
        Color color;
        setColorGrid(&(color.components), GRID_EXPLORED_OBS);
        pos_obs[tmp] = make_float4(x_check,y_check,GRID_EXPLORED_OBS,color.c);
        /*printf("obs = array[%d]=%f\n",tmp, pos_obs[tmp].w);*/

        is_obs_encountered = true;
        break;
      }
    }
  }
}

__device__ int occupancySub2Ind(float x, float y, Parameters p) {
  float ws_2 = (float)(p.world_size) / 2.0f;
  float ws_10 = (float)(p.world_size) * 10.0f;
  // Return -1 if the coordinates to check are outside the world boundaries;
  // else, return the index in the occupancy grid for these coordinates
  if (x < -ws_2 || x > ws_2 || y < -ws_2 || y > ws_2) {
    /*return true;*/
    return -1;
  }
  else {
    // Get the 1d index for the occupancy array from the x and y coordinates
    uint x_component = (uint)((x + ws_2) * 10.0f);
    uint y_component = (uint)(floorf(y + ws_2) * ws_10 * 10.0f);
    uint index = x_component + y_component;
    /*return occupancy[index];*/
    return index;
  }
}

/// Set color for swarm robot members
__device__ void setColorSwarm(uchar4* color, int mode, bool is_ap, uint i, 
  Parameters p, bool is_obs_encountered) {

  /*draw non-leaders in (255,255,255)*/
  /*draw leaders in (255,0,0)*/
  /*draw articulation points in (0,200,0)*/
  /*draw noise robots in (100,100,100)*/

  /// uchar is an integer between 0 to 255
  /// https://stackoverflow.com/questions/75191/what-is-an-unsigned-charhttps://stackoverflow.com/questions/75191/what-is-an-unsigned-char
  /// Here uchar4 is a tuple of 4 integers in that range
  /// uchar4* is an array of uchar4

  /// ap = articulation points

  if (mode == MODE_LEADER && p.show_leaders) {
    if (p.highlight_leaders) {
      (is_ap && p.show_ap) ? 
                *color = make_uchar4(0, 200, 0, 255) : 
                *color = make_uchar4(255, 0, 0, 255);
    }
    else {
      (is_ap && p.show_ap) ? 
                *color = make_uchar4(0, 200, 0, 255) : 
                *color = make_uchar4(255, 255, 255, 255);
    }
  }
  else if (mode != MODE_LEADER && p.show_non_leaders) {
    if (mode > MODE_LEADER) {
      (is_ap && p.show_ap) ? 
                *color = make_uchar4(0, 200, 0, 255) : 
                *color = make_uchar4(255, 255, 255, 255);
    }
    else {
      *color = make_uchar4(100, 100, 100, 255);
    }
  }
  else {
    *color = make_uchar4(0, 0, 0, 0);
  }

  /// draw extra color for obstacle encounted robots
  if (p.highlight_pioneers == true && is_obs_encountered == true) {
    *color = make_uchar4(255, 255, 0, 255);
  }
}

/// Set color for grid
__device__ void setColorGrid(uchar4* color, int grid) {

  /*draw explored obstacle in (255,255,0)*/
  /*draw others in (0,0,0)*/
  if (grid == GRID_EXPLORED_OBS) {
    *color = make_uchar4(255, 255, 0, 255);
  }
  else {
    printf("WTF\n");
  }
  /*else if (grid == GRID_UNEXPLORED) {*/
    /**color = make_uchar4(0, 255, 0, 255);*/
  /*}*/
  /*else if (grid == GRID_EXPLORED_FREE) {*/
    /**color = make_uchar4(0, 0, 100, 255);*/
  /*}*/
}



__device__ float euclidean(float2 vector)
{
  return sqrtf(powf(vector.x, 2.0f) + powf(vector.y, 2.0f));
}

__device__ void rescale(float2* vel, float value, bool is_value_limit)
{
  // Determine the scalar of the vector
  float scalar = euclidean(*vel);

  // Normalize the vector if the value given is the limit and the scalar is  
  // above that, or if the value given is not a limit
  if (scalar != 0.0f && ((is_value_limit && scalar > value) || !is_value_limit)) {
    float factor = value / scalar;
    vel->x *= factor;
    vel->y *= factor;
  }
}

__device__ void normalizeAngle(float* angle)
{
  while (*angle > PI) {
    *angle -= 2.0f * PI;
  }
  while (*angle <= -PI) {
    *angle += 2.0f * PI;
  }
}

__device__ void capAngularVelocity(float2 old, float2* goal, float max)
{
  // Get the magnitude of each vector
  float norm_old = euclidean(old);
  float norm_goal = euclidean(*goal);

  float old_angle = atan2f(old.y, old.x);
  float goal_angle = atan2f(goal->y, goal->x);

  // Get the angle from the old to new goal vector
  float angle = goal_angle - old_angle;
  normalizeAngle(&angle);

  // If the angle is greater than the maximum angular velocity, cap the 
  // angular velocity to this maximum
  if (fabsf(angle) > max && norm_old != 0.0f && norm_goal != 0.0f) {
    (angle < 0.0f) ? old_angle -= max : old_angle += max;
    goal->x = norm_goal * cosf(old_angle);
    goal->y = norm_goal * sinf(old_angle);
  }
}
